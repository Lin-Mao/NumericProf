#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define CUDA_SAFECALL(call)                                                 \
    {                                                                       \
        call;                                                               \
        hipError_t err = hipGetLastError();                                 \
        if (hipSuccess != err) {                                           \
            fprintf(                                                        \
                stderr,                                                     \
                "Cuda error in function '%s' file '%s' in line %i : %s.\n", \
                #call, __FILE__, __LINE__, hipGetErrorString(err));        \
            fflush(stderr);                                                 \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }


// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(__half *a, __half *b, __half *c, int n) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n) c[id] = __hmul(a[id], b[id]);
}

int main(int argc, char *argv[]) {
    // Size of vectors
    int n = 100000;
    if (argc > 1) n = atoi(argv[1]);

    // Host input vectors
    __half *h_a;
    __half *h_b;
    // Host output vector
    __half *h_c;

    // Device input vectors
    __half *d_a;
    __half *d_b;
    // Device output vector
    __half *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(__half);

    // Allocate memory for each vector on host
    h_a = (__half *)malloc(bytes);
    h_b = (__half *)malloc(bytes);
    h_c = (__half *)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    // Initialize vectors on host
    for (i = 0; i < n; i++) {
        h_a[i] = __float2half(1);
        h_b[i] = __float2half(1);
        h_c[i] = __float2half(1);
    }

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n / blockSize);

    // Execute the kernel
    CUDA_SAFECALL((vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));

    // Copy array back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Sum up vector c and print result divided by n, this should equal 1 within
    // error
    float sum = 0;
    for (i = 0; i < n; i++) sum += float(h_c[i]);
    printf("Final sum = %f; sum/n = %f (should be ~1)\n", sum, sum / n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
